#include "hip/hip_runtime.h"
#include"FFT2CUDA2.h"
#include"math.h"

texture<float,2,hipReadModeElementType>texRefR;//ʵ���������ϵ����
texture<float,2,hipReadModeElementType>texRefI;//�鲿�������ϵ����
__global__ void FFT2Cudaa(float*DataInR,float*DataInI,float*DataOutR,float*DataOutI,float*WnR,float*WnI,int N)
{
__shared__ float sdataR[64],sdataI[64];
int ttid=threadIdx.x;
if(ttid<N)
{
sdataR[ttid]=DataInR[ttid];
sdataI[ttid]=0;
sdataR[ttid+N/2]=DataInR[ttid+N/2];
sdataI[ttid+N/2]=0;
__syncthreads();
if(ttid<N/2)
    {
    int p,q;
    float WR,WI,XpR,XpI,XqWnR,XqWnI;
    int stage1=0;float stage2=0;
    for(int Nd=1;Nd<N;Nd=Nd*2)
        {
        p=rintf((ttid/Nd)-0.5)*Nd*2+ttid%Nd;
        q=p+Nd;
        WR=(float)(WnR[ttid+32*(stage1++)]);
        WI=(float)(WnI[ttid+32*(stage2++)]);
        XqWnR=sdataR[q]*WR-sdataI[q]*WI;//��������
        XqWnI=sdataR[q]*WI+sdataI[q]*WR;
        XpR=sdataR[p];
        XpI=sdataI[p];
        sdataR[p]=XpR+XqWnR;
        sdataI[p]=XpI+XqWnI;
        sdataR[q]=XpR-XqWnR;
        sdataI[q]=XpI-XqWnI;
        __syncthreads();
        }
    DataOutR[p]=sdataR[p];
    DataOutI[p]=sdataI[p];
    DataOutR[q]=sdataR[q];
    DataOutI[q]=sdataI[q];
    }
}
}
void FFt2Cudae2(float*DataInR,float*DataInI,float*DataOutR,float*DataOutI,float*wnR,float*wnI,int Nn) 
{
   
    float*DeviceSrcR,*DeviceSrcI,*DeviceDstR,*DeviceDstI,*DeviceWnR,*DeviceWnI;

    hipMalloc(&DeviceSrcR,sizeof(float)*Nn);
    hipMalloc(&DeviceSrcI,sizeof(float)*Nn);
    hipMalloc(&DeviceDstR,sizeof(float)*Nn);
    hipMalloc(&DeviceDstI,sizeof(float)*Nn);
    hipMalloc(&DeviceWnR,sizeof(float)*Nn/2);
    hipMalloc(&DeviceWnI,sizeof(float)*Nn/2);
    
    hipMemcpy(DeviceSrcR,DataInR,sizeof(float)*64,hipMemcpyHostToDevice);//���ݴ���Device
    hipMemcpy(DeviceSrcI,DataInI,sizeof(float)*64,hipMemcpyHostToDevice);
    hipMemcpy(DeviceWnR,wnR,sizeof(float)*32,hipMemcpyHostToDevice);//WN����Device
    hipMemcpy(DeviceWnI,wnI,sizeof(float)*32,hipMemcpyHostToDevice);

    hipMemset(DeviceDstR,0,sizeof(float)*Nn);
    hipMemset(DeviceDstI,0,sizeof(float)*Nn);

    dim3 blockSize(32,1,1);
    dim3 gridSize(1,1,1);

    const size_t smemSize=128*sizeof(float);
    FFT2Cudaa<<<gridSize,blockSize,smemSize>>>(DeviceSrcR,DeviceSrcI,DeviceDstR,DeviceDstI,DeviceWnR,DeviceWnI,Nn);
    
    hipMemcpy(DataOutR,DeviceDstR,sizeof(float)*Nn,hipMemcpyDeviceToHost);
    hipMemcpy(DataOutI,DeviceDstI,sizeof(float)*Nn,hipMemcpyDeviceToHost);


    hipFree(DeviceSrcR);
    hipFree(DeviceDstR);
    hipFree(DeviceSrcI);
    hipFree(DeviceDstI);
    hipFree(DeviceWnR);
    hipFree(DeviceWnI);
}